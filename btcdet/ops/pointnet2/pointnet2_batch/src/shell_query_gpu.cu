#include "hip/hip_runtime.h"
/*
batch version of shell query, modified from the original implementation of official PointNet++ codes.
Written by Shaoshuai Shi
All Rights Reserved 2018.
*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "shell_query_gpu.h"
#include "cuda_utils.h"


__global__ void shell_query_kernel_fast(int b, int n, int m, float inner_radius, float outer_radius, int nsample,
    const float *__restrict__ new_xyz, const float *__restrict__ xyz, int *__restrict__ idx) {
    // new_xyz: (B, M, 3)
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, M, nsample)
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= m) return;

    new_xyz += bs_idx * m * 3 + pt_idx * 3;
    xyz += bs_idx * n * 3;
    idx += bs_idx * m * nsample + pt_idx * nsample;

    float inner_radius2 = inner_radius * inner_radius;
    float outer_radius2 = outer_radius * outer_radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    int cnt = 0;
    for (int k = 0; k < n; ++k) {
        float x = xyz[k * 3 + 0];
        float y = xyz[k * 3 + 1];
        float z = xyz[k * 3 + 2];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
        if (d2 >= inner_radius2 && d2 < outer_radius2){
            if (cnt == 0){
                for (int l = 0; l < nsample; ++l) {
                    idx[l] = k;
                }
            }
            idx[cnt] = k;
            ++cnt;
            if (cnt >= nsample) break;
        }
    }
}


void shell_query_kernel_launcher_fast(int b, int n, int m, float inner_radius, float outer_radius, int nsample, \
    const float *new_xyz, const float *xyz, int *idx) {
    // new_xyz: (B, M, 3)
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, M, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    shell_query_kernel_fast<<<blocks, threads>>>(b, n, m, inner_radius, outer_radius, nsample, new_xyz, xyz, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
